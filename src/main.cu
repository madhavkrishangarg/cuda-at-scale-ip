#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

__global__ void rgbToGrayscale(unsigned char* d_input, unsigned char* d_output, int width, int height, int channels) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = (y * width + x) * channels;

    if (x < width && y < height) {
        unsigned char r = d_input[idx];
        unsigned char g = d_input[idx + 1];
        unsigned char b = d_input[idx + 2];
        d_output[y * width + x] = 0.299f * r + 0.587f * g + 0.114f * b;
    }
}

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <image_path>" << std::endl;
        return -1;
    }

    std::string imagePath = argv[1];
    cv::Mat image = cv::imread(imagePath, cv::IMREAD_COLOR);
    if (image.empty()) {
        std::cerr << "Error: Could not open or find the image!" << std::endl;
        return -1;
    }

    int width = image.cols;
    int height = image.rows;
    int channels = image.channels();

    cv::Mat grayImage(height, width, CV_8UC1);

    unsigned char *d_input, *d_output;
    size_t imageSize = width * height * channels * sizeof(unsigned char);
    size_t grayImageSize = width * height * sizeof(unsigned char);

    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, grayImageSize);

    hipMemcpy(d_input, image.data, imageSize, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    rgbToGrayscale<<<gridSize, blockSize>>>(d_input, d_output, width, height, channels);

    hipMemcpy(grayImage.data, d_output, grayImageSize, hipMemcpyDeviceToHost);

    cv::imwrite("output.png", grayImage);

    hipFree(d_input);
    hipFree(d_output);

    return 0;
} 